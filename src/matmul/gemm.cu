#include "hip/hip_runtime.h"
#include "manifest.h"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"

namespace cutlass {
namespace library {
void initialize_all_gemm_operations(Manifest &manifest);
} // namespace library
} // namespace cutlass

int main() {
  Manifest manifest;
  cutlass::library::initialize_all_gemm_operations(manifest);

  manifest.template profile<__half, __half, float, float>(
      1024, 1024, 1024, LayoutEnum::ColumnMajor, LayoutEnum::ColumnMajor,
      LayoutEnum::RowMajor);
  return 0;
}